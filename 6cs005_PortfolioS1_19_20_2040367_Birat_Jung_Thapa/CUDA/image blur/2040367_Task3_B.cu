#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

/********
Compile with nvcc 2040367_Task3_B.cu lodepng.cpp -o task3b

             ./task3b
*********/

__global__ void blur_image(unsigned char * gpu_imageOuput, unsigned char * gpu_imageInput,int width,int height){

    int counter=0;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    
    int i=blockIdx.x;
    int j=threadIdx.x;


    float t_r=0;
	float t_g=0;
	float t_b=0;
    float t_a=0;
    float s=1;

    if(i+1 && j-1){

        // int pos= idx/2-2;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x-1;
        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];
        
        counter++;



    }

    if(j+1){

        // int pos= idx/2-2;

        int pos=blockDim.x * (blockIdx.x) + threadIdx.x+1;

        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];

        counter++;
    }

    if(i+1 && j+1){

        // int pos= idx/2+1;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x+1;


        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];

        counter++;


    }

    if(i+1){
        // int pos= idx+1;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x;

        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];

        counter++;



    }

    if(j-1){

        // int pos= idx*2-2;
        int pos=blockDim.x * (blockIdx.x) + threadIdx.x-1;

        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];

        counter++;




    }

    if(i-1){

        // int pos= idx-1;
        int pos=blockDim.x * (blockIdx.x-1) + threadIdx.x;

        int pixel = pos*4;

        // t_r=s*gpu_imageInput[idx*4];
        // t_g=s*gpu_imageInput[idx*4+1];
        // t_b=s*gpu_imageInput[idx*4+2];
        // t_a=s*gpu_imageInput[idx*4+3];

        t_r += s*gpu_imageInput[pixel];
        t_g += s*gpu_imageInput[1+pixel];
        t_b += s*gpu_imageInput[2+pixel];
        t_a += s*gpu_imageInput[3+pixel];

        counter++;


    }
    
    int current_pixel=idx*4;

    gpu_imageOuput[current_pixel]=(int)t_r/counter;
    gpu_imageOuput[1+current_pixel]=(int)t_g/counter;
    gpu_imageOuput[2+current_pixel]=(int)t_b/counter;
    gpu_imageOuput[3+current_pixel]=gpu_imageInput[3+current_pixel];


}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}

int main(int argc, char **argv){
struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);
	
	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "image.png";
	const char* newFileName = "blur.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	blur_image<<<height, width>>>(d_out, d_in,width,height);


	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);
	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 
	
	return 0;
}

